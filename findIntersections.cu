#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <iterator>

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/count.h>
#include <thrust/functional.h>


//__device__ __host__ bool moreThan5(int amount) {
//    return amount > 5;
//}


/**
 * Generates an array of ints representing the height (in number of layers) which each triangle spans
 * @param zs
 * @param layersInTris
 * @param numberOfTris
 * @return
 */
__global__ void layersInEachTriangle(float* zs, int* layersInTris, int numberOfTris, const float lH) {
    // Represents the triangle we're checking currently
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    float z0 = zs[gtid*3];
    float z1 = zs[gtid*3+1];
    float z2 = zs[gtid*3+2];
    int layersContained;
    if(gtid < numberOfTris) {
        float z_max = max(max(z0, z1), z2);
        float z_min = min(min(z0, z1), z2);
        layersContained = ceil(z_max/lH) - floor(z_min / lH);
        if (layersContained*lH + z_min == z_max)
            layersContained++;
        layersInTris[gtid] = layersContained;
    }

    //TODO: Handle boundary case of triangles which intersect layer at a point or are completely coplanar
}

//__device__ float dot(float* xs, float* ys, float* zs, int i0, int i1) {
//
//}

/**
 *
 * @param x0
 * @param x1
 * @param y0
 * @param y1
 * @param z0
 * @param z1
 * @param zp z of the plane
 * @param x_r result of the intersection
 * @param y_r result of the intersection
 * @return
 */
__device__ void get_intersection(float x0, float x1, float y0, float y1, float z0, float z1, float zp, float* x_r, float* y_r) {


    //TODO: Put this check outside and handle by putting both points into the segment list. This case represents a planar line segment
    float denom = (z1 - z0);

    printf("z0: %f, z1: %f\n", z0, z1);

    if(denom == 0) {
        x_r = nullptr;
        y_r = nullptr;
        return;
    }

    float t = (zp - z0) / denom;

    if(t < 0 || t > 1) {
//        printf("84\n");
        x_r = nullptr;
        y_r = nullptr;
        return;
    }

    printf("t: %f\n", t);
    *x_r = x0 + t * (x1 - x0);
    *y_r = y0 + t * (y1 - y0);
    printf("Intersection of %f,%f,%f -> %f, %f, %f with z=%f: %f, %f\n", x0, y0, z0, x1, y1, z1, zp, *x_r, *y_r);
    return;
}

__global__ void calculateIntersections(float* xs, float* ys, float* zs, int* layersInTri, int* startIndexInSegments, float* seg_x, float* seg_y, float* seg_l, const float lH, const int n) {
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    if(gtid < n) {
        int stri = gtid*3;
        float x0 = xs[stri];
        float y0 = ys[stri];
        float z0 = zs[stri];
        float x1 = xs[stri + 1];
        float y1 = ys[stri + 1];
        float z1 = zs[stri + 1];
        float x2 = xs[stri + 2];
        float y2 = ys[stri + 2];
        float z2 = zs[stri + 2];
        float bottomLayer = floor(min(min(z0, z1), z2) / lH);

        // Iterate through layers
        int layer; float zp;
        float* x_r = (float*)malloc(sizeof(float));
        float* y_r = (float*)malloc(sizeof(float));
        int intersectionsFound;

        printf("layersInTri[gtid]:%d\n", layersInTri[gtid]);
        for(int i = 0; i < layersInTri[gtid]; i++) {
            layer = bottomLayer + i;
            zp = layer * lH;

            intersectionsFound = 0;

            printf("gtid:%d\n", gtid);
            printf("zp: %f\n",zp);

            get_intersection(x0, x1, y0, y1, z0, z1, zp, x_r, y_r);
            if(x_r != nullptr){
                seg_x[startIndexInSegments[gtid]*2] = *x_r;
                seg_y[startIndexInSegments[gtid]*2] = *y_r;
                intersectionsFound++;
            }

            get_intersection(x1, x2, y1, y2, z1, z2, zp, x_r, y_r);
            printf("137\n");
            if(x_r != nullptr){
                seg_x[startIndexInSegments[gtid]*2+intersectionsFound] = *x_r;
                seg_y[startIndexInSegments[gtid]*2+intersectionsFound] = *y_r;
                intersectionsFound++;
            }

            get_intersection(x2, x0, y2, y0, z2, z0, zp, x_r, y_r);
            if(x_r != nullptr){
                seg_x[startIndexInSegments[gtid]*2+intersectionsFound] = *x_r;
                seg_y[startIndexInSegments[gtid]*2+intersectionsFound] = *y_r;
                intersectionsFound++;
            }
            // TODO: Handle boundary cases for planar triangles and tangentially intersecting triangles
        }
    }
//    for(int i=0; i < 3; i++) {
//        sx[stri + i]
//    }

}

// simple routine to print contents of a vector
template <typename Vector>
void print_vector(const std::string& name, const Vector& v)
{
    typedef typename Vector::value_type T;
    std::cout << "  " << std::setw(20) << name << "  ";
    thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
    std::cout << std::endl;
}

int main(int argc, char* argv[]) {

    float time = 0.f;

    const int n = 1;
    const int N = n*3;
    const float layerHeight = .5;


//    float x[N] = {0.,  0., 1., 0., 1., 1., 0., 0., 0.};
//    float y[N] = {0.,  0., 0., 0., 0., 0., 0., 1., 1.};
//    float z[N] = {.25, 1.25, 1.25, 0.25, 0.25, 1.25, 1.25, 2.6, 2.6};
    float x[N] = {0.,  0., 1.};
    float y[N] = {0.,  0., 0.};
    float z[N] = {.25, 1.25, 1.25};


    // Timing things
    hipEvent_t startEvent_inc, stopEvent_inc;
    hipEventCreate(&startEvent_inc);
    hipEventCreate(&stopEvent_inc);
    hipEventRecord(startEvent_inc,0);

    thrust::device_vector<float> dx(N, 0);
    thrust::device_vector<float> dy(N, 0);
    thrust::device_vector<float> dz(N, 0);
    thrust::copy(x, x+N, dx.begin());
    thrust::copy(y, y+N, dy.begin());
    thrust::copy(z, z+N, dz.begin());

    print_vector("x", dx);

    thrust::device_vector<int> layersInTris(n, 0);
    int*  layersInTris_p = thrust::raw_pointer_cast( &layersInTris[0] );
    float*  z_p = thrust::raw_pointer_cast( &dz[0] );
    layersInEachTriangle<<<2, 8>>>(z_p, layersInTris_p, n, layerHeight);

    print_vector("layersInTris", layersInTris);

    thrust::device_vector<int> intersectionSegmentsIndexStart(n, 0);
    thrust::inclusive_scan(layersInTris.begin(), layersInTris.end(), intersectionSegmentsIndexStart.begin());

    print_vector("intersectionSegmentsIndexStart", intersectionSegmentsIndexStart);

    int totalIntersections = intersectionSegmentsIndexStart[intersectionSegmentsIndexStart.size()-1];

    printf("totalIntersections: %d\n", totalIntersections);

    // Intersection segment coordinate arrays
    thrust::device_vector<float> iscx(totalIntersections*2, 0);
    thrust::device_vector<float> iscy(totalIntersections*2, 0);
    thrust::device_vector<float> iscl(totalIntersections*2, 0);
    float* iscx_p = thrust::raw_pointer_cast( &iscx[0] );
    float* iscy_p = thrust::raw_pointer_cast( &iscy[0] );
    float* iscl_p = thrust::raw_pointer_cast( &iscl[0] );
    float* x_p = thrust::raw_pointer_cast( &x[0] );
    float* y_p = thrust::raw_pointer_cast( &y[0] );
    int* intersectionSegmentsIndexStart_p = thrust::raw_pointer_cast( &intersectionSegmentsIndexStart[0]);

    calculateIntersections<<<2, 8>>>(x_p, y_p, z_p, layersInTris_p, intersectionSegmentsIndexStart_p, iscx_p, iscy_p, iscl_p, layerHeight, n);

//    print_vector("iscx", iscx);
//    print_vector("iscy", iscy);



//    thrust::equal_to<int> binary_pred;
//    thrust::plus<int> binary_op;
//    thrust::maximum<int> max_fn;
//
//    int day_out[N];
//    int measurement_maxes[N];
//    ////Part A:
//    thrust::reduce_by_key(day, day+N, measurement, day_out, measurement_maxes, binary_pred, max_fn);
//
//    int numSites = thrust::count_if(measurement_maxes, measurement_maxes+11, moreThan5);
//
//    printf("%d\n", numSites);
//
//    ////Part B:
//    thrust::sort_by_key(site, site + N, measurement);
//
//
//    thrust::reduce_by_key(thrust::host,
//                          site, site + N, measurement,
//                          site, measurement,
//                          binary_pred, binary_op);
//
//    for(int i=0; i<5; i++) {
//        printf("%d ", measurement[i]);
//    }
//    printf("\n");
//

    // Finish timing
    hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
    hipEventSynchronize(stopEvent_inc);
    hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);

    // END MY CODE

    //printf("%d\n%f\n%f\n\n",N,cuda_out[N-1],time);

    //free resources
    //free(in); free(out); free(cuda_out);
    return 0;
}